#include "hip/hip_runtime.h"
#include <optix.h>
#include "random.h"
#include "LaunchParams7.h" // our launch params
#include "vec_math.h" // NVIDIAs math utils


extern "C" {
    __constant__ LaunchParams optixLaunchParams;
}
//  a single ray type
enum { PHONG=0, SHADOW, RAY_TYPE_COUNT };

struct colorPRD {
    float3 color;
    unsigned int seed;
} ;

struct shadowPRD {
    float shadowAtt;
    unsigned int seed;
} ;

// -------------------------------------------------------
// closest hit computes color based lolely on the triangle normal
extern "C" __global__ void __closesthit__radiance() {
    
    colorPRD &prd = *(colorPRD*)getPRD<colorPRD>();
    
    const TriangleMeshSBTData &sbtData
    = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  
    
    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];
    
    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;
    
    // compute normal
    const float4 n
    = (1.f-u-v) * sbtData.vertexD.normal[index.x]
    +         u * sbtData.vertexD.normal[index.y]
    +         v * sbtData.vertexD.normal[index.z];
    float3 nn = normalize(make_float3(n));
    
    // intersection position
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();
    
    // Ambient Occlusion
    int numRays = 4;
    float ambientOcclusion = 0;
    shadowPRD AOPRD;
    AOPRD.shadowAtt = 1.0f;
    AOPRD.seed = prd.seed;
    
    uint32_t u0, u1;
    packPointer(&AOPRD, u0, u1);
    
    for (int i = 0; i < numRays; ++i) {
        const float z1 = rnd(prd.seed);
        const float z2 = rnd(prd.seed);
        
        float3 rayDir, p;
        
        // Uniform hemisphere sampling
        // p.y = z1;
        // float k = sqrt(1-pow(p.y, 2));
        // float beta = 2 * M_PIf * z2;
        // p.x = k * sin(beta);
        // p.z = k * cos(beta);
        
        // cosine weighted hemisphere sampling
        float r = sqrt(z1);
        float beta = 2 * M_PIf * z2;
        p.x = r * sin(beta);
        p.z = r * cos(beta);
        p.y = sqrt(1-pow(r,2));

        // Converting from sampling to world space
        float3 t;
        float3 c1 = cross(nn, make_float3(0.0, 0.0, 1.0));
        float3 c2 = cross(nn, make_float3(0.0, 1.0, 0.0));
        
        if( length(c1) > length(c2) ) {
            t = c1;
        }
        else {
            t = c2;
        }
        normalize(t);
        float3 b = cross(t, nn);
        rayDir = p.x * b + p.y * nn + p.z * t;
        
        // trace ambient occlusion
        optixTrace(optixLaunchParams.traversable,
                   pos,
                   rayDir,
                   0.00001f,    // tmin
                   0.5f,        // tmax
                   0.0f,        // rayTime
                   OptixVisibilityMask( 255 ),
                   OPTIX_RAY_FLAG_NONE,
                   SHADOW,             // SBT offset
                   RAY_TYPE_COUNT,     // SBT stride
                   SHADOW,             // missSBTIndex 
                   u0, u1);
        
        ambientOcclusion += AOPRD.shadowAtt;
    }
    
    prd.color = make_float3(ambientOcclusion / numRays);
}


// any hit to ignore intersections with back facing geometry
extern "C" __global__ void __anyhit__radiance() {
    
}


// miss sets the background color
extern "C" __global__ void __miss__radiance() {
    
    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(0.0f, 0.0f, 1.0f);
}


// -----------------------------------------------
// Shadow rays

extern "C" __global__ void __closesthit__shadow() {
    float &prd = *(float*)getPRD<float>();
    prd = 0.0f;
}


// any hit for shadows
extern "C" __global__ void __anyhit__shadow() {
    
}


// miss for shadows
extern "C" __global__ void __miss__shadow() {
    float &prd = *(float*)getPRD<float>();
    // set blue as background color
    prd = 1.0f;
}


// -----------------------------------------------
// Light material


extern "C" __global__ void __closesthit__light() {
    float3 &prd = *(float3*)getPRD<float3>();
    prd = make_float3(1.0f, 1.0f, 1.0f);
}


extern "C" __global__ void __anyhit__light() {
}


extern "C" __global__ void __miss__light() {
}


extern "C" __global__ void __closesthit__light_shadow() {
    float &prd = *(float*)getPRD<float>();
    prd = 1.0f;
}


// any hit to ignore intersections based on alpha transparency
extern "C" __global__ void __anyhit__light_shadow() {
}


// miss sets the background color
extern "C" __global__ void __miss__light_shadow() {
}


// -----------------------------------------------
// Metal Phong rays

extern "C" __global__ void __closesthit__phong_metal() {
    // ray payload
    colorPRD &prd = *(colorPRD*)getPRD<colorPRD>();
    
    const TriangleMeshSBTData &sbtData
    = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  
    
    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];
    
    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;
    
    // compute normal
    const float4 n
    = (1.f-u-v) * sbtData.vertexD.normal[index.x]
    +         u * sbtData.vertexD.normal[index.y]
    +         v * sbtData.vertexD.normal[index.z];
    
    float3 normal = normalize(make_float3(n));
       
    colorPRD afterPRD;
    afterPRD.color = make_float3(1.0f);
    afterPRD.seed = prd.seed;
    uint32_t u0, u1;
    packPointer( &afterPRD, u0, u1 );  
    
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();
    
    float3 rayDir = reflect(optixGetWorldRayDirection(), normal);
    optixTrace(optixLaunchParams.traversable,
               pos,
               rayDir,
               0.04f,    // tmin is high to void self-intersection
               1e20f,  // tmax
               0.0f,   // rayTime
               OptixVisibilityMask( 255 ),
               OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
               PHONG,             // SBT offset
               RAY_TYPE_COUNT,     // SBT stride
               PHONG,             // missSBTIndex 
               u0, u1 );
    
    prd.color = make_float3(0.8,0.8,0.8) * afterPRD.color;
}



// -----------------------------------------------
// Glass Phong rays

SUTIL_INLINE SUTIL_HOSTDEVICE float3 refract(const float3& i, const float3& n, const float eta) {
    
    float k = 1.0 - eta * eta * (1.0 - dot(n, i) * dot(n, i));
    if (k < 0.0)
        return make_float3(0.0f);
    else
        return (eta * i - (eta * dot(n, i) + sqrt(k)) * n);
}


extern "C" __global__ void __closesthit__phong_glass() {
    // ray payload 
    colorPRD &prd = *(colorPRD*)getPRD<colorPRD>();
    
    const TriangleMeshSBTData &sbtData
    = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  
    
    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];
    
    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;
    
    // compute normal
    const float4 n
    = (1.f-u-v) * sbtData.vertexD.normal[index.x]
    +         u * sbtData.vertexD.normal[index.y]
    +         v * sbtData.vertexD.normal[index.z];
    
    float3 normal = normalize(make_float3(n));
    const float3 normRayDir = optixGetWorldRayDirection();
    
    // new ray direction
    float3 rayDir;
    // entering glass
    float dotP;
    if (dot(normRayDir, normal) < 0) {
        dotP = dot(normRayDir, -normal);
        rayDir = refract(normRayDir, normal, 0.66);
    }
    // exiting glass
    else {
        dotP = 0;
        rayDir = refract(normRayDir, -normal, 1.5);
    }
    
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();
    
    colorPRD refractPRD;
    refractPRD.color = make_float3(0.0f);
    refractPRD.seed = prd.seed;
    uint32_t u0, u1;
    packPointer( &refractPRD, u0, u1 );  
    
    if (length(rayDir) > 0)
        optixTrace(optixLaunchParams.traversable,
                   pos,
                   rayDir,
                   0.00001f,    // tmin
                   1e20f,       // tmax
                   0.0f,        // rayTime
                   OptixVisibilityMask( 255 ),
                   OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
                   PHONG,               // SBT offset
                   RAY_TYPE_COUNT,      // SBT stride
                   PHONG,               // missSBTIndex 
                   u0, u1 );
        
        
    colorPRD reflectPRD;
    reflectPRD.color = make_float3(0.0f);
    reflectPRD.seed = prd.seed;
    if (dotP > 0) {
        float3 reflectDir = reflect(normRayDir, normal);        
        packPointer( &reflectPRD, u0, u1 );  
        optixTrace(optixLaunchParams.traversable,
                    pos,
                    reflectDir,
                    0.00001f,    // tmin
                    1e20f,       // tmax
                    0.0f,        // rayTime
                    OptixVisibilityMask( 255 ),
                    OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
                    PHONG,               // SBT offset
                    RAY_TYPE_COUNT,      // SBT stride
                    PHONG,               // missSBTIndex 
                    u0, u1 );
        float r0 = (1.5f - 1.0f)/(1.5f + 1.0f);
        r0 = r0*r0 + (1-r0*r0) * pow(1-dotP,5);
        prd.color =  refractPRD.color * (1-r0) + r0*reflectPRD.color;
    }
    else
        prd.color =  refractPRD.color ;
}



extern "C" __global__ void __anyhit__phong_glass() {
    
}


// miss sets the background color
extern "C" __global__ void __miss__phong_glass() {
    
    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(0.0f, 0.0f, 1.0f);
}



// -----------------------------------------------
// Glass Shadow rays

extern "C" __global__ void __closesthit__shadow_glass() {
    
    // ray payload
    float afterPRD = 1.0f;
    uint32_t u0, u1;
    packPointer( &afterPRD, u0, u1 );  
    
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();
    
    // trace primary ray
    optixTrace(optixLaunchParams.traversable,
               pos,
               optixGetWorldRayDirection(),
               0.001f,    // tmin
               1e20f,     // tmax
               0.0f,      // rayTime
               OptixVisibilityMask( 255 ),
               OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
               SHADOW,              // SBT offset
               RAY_TYPE_COUNT,      // SBT stride
               SHADOW,              // missSBTIndex 
               u0, u1 );
    
    float &prd = *(float*)getPRD<float>();
    prd = 0.95f * afterPRD;
}


// any hit for shadows
extern "C" __global__ void __anyhit__shadow_glass() {
    
}


// miss for shadows
extern "C" __global__ void __miss__shadow_glass() {
    float &prd = *(float*)getPRD<float>();
    // set blue as background color
    prd = 1.0f;
}

// -----------------------------------------------
// Primary Rays

extern "C" __global__ void __raygen__renderFrame() {
    
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto &camera = optixLaunchParams.camera;  
    
    
    if (optixLaunchParams.frame.frame == 0 && ix == 0 && iy == 0) {
        
        // print info to console
        printf("===========================================\n");
        printf("Nau Ray-Tracing Debug\n");
        const float4 &ld = optixLaunchParams.global->lightPos;
        printf("LightPos: %f, %f %f %f\n", ld.x,ld.y,ld.z,ld.w);
        printf("Launch dim: %u %u\n", optixGetLaunchDimensions().x, optixGetLaunchDimensions().y);
        printf("Rays per pixel squared: %d \n", optixLaunchParams.frame.raysPerPixel);
        printf("===========================================\n");
    }
    
    
    // ray payload
    colorPRD pixelColorPRD;
    pixelColorPRD.color = make_float3(1.f);
    uint32_t u0, u1;
    packPointer( &pixelColorPRD, u0, u1 );  
    
    float raysPerPixel = float(optixLaunchParams.frame.raysPerPixel);
    // half pixel
    float2 delta = make_float2(1.0f/raysPerPixel, 1.0f/raysPerPixel);
    
    // compute ray direction
    // normalized screen plane position, in [-1, 1]^2
    
    float red = 0.0f, blue = 0.0f, green = 0.0f;
    for (int i = 0; i < raysPerPixel; ++i) {
        for (int j = 0; j < raysPerPixel; ++j) {
            float2 subpixel_jitter;
            uint32_t seed = tea<4>( ix * optixGetLaunchDimensions().x + iy, i*raysPerPixel + j );
            pixelColorPRD.seed = seed;

            subpixel_jitter = make_float2( rnd( seed )-0.5f, rnd( seed )-0.5f );
            
            const float2 screen(make_float2(ix + subpixel_jitter.x, iy + subpixel_jitter.y)
            / make_float2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y) * 2.0 - 1.0);
            
            // note: nau already takes into account the field of view and ratio when computing 
            // camera horizontal and vertival
            float3 rayDir = normalize(camera.direction
            + (screen.x ) * camera.horizontal
            + (screen.y ) * camera.vertical);
            
            // trace primary ray
            optixTrace(optixLaunchParams.traversable,
                       camera.position,
                       rayDir,
                       0.f,    // tmin
                       1e20f,  // tmax
                       0.0f,   // rayTime
                       OptixVisibilityMask( 255 ),
                       OPTIX_RAY_FLAG_NONE, //,OPTIX_RAY_FLAG_DISABLE_ANYHIT
                       PHONG,               // SBT offset
                       RAY_TYPE_COUNT,      // SBT stride
                       PHONG,               // missSBTIndex 
                       u0, u1 );
            
            red += pixelColorPRD.color.x / (raysPerPixel*raysPerPixel);
            green += pixelColorPRD.color.y / (raysPerPixel*raysPerPixel);
            blue += pixelColorPRD.color.z / (raysPerPixel*raysPerPixel);
        }
    }
    
    //convert float (0-1) to int (0-255)
    const int r = int(255.0f*red);
    const int g = int(255.0f*green);
    const int b = int(255.0f*blue);
    // convert to 32-bit rgba value 
    const uint32_t rgba = 0xff000000
    | (r<<0) | (g<<8) | (b<<16);
    // compute index
    const uint32_t fbIndex = ix + iy*optixGetLaunchDimensions().x;
    // write to output buffer
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}


