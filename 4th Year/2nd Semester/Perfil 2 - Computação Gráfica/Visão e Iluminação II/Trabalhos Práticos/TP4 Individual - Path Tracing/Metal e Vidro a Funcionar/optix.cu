#include "hip/hip_runtime.h"
 // Our Launch Params
#include "optixParams.h"

extern "C"
{
    __constant__ LaunchParams optixLaunchParams;
}

// Ray Types
enum { RAIDANCE=0, SHADOW, RAY_TYPE_COUNT };

struct ShadowPRD
{
    float       shadowAtt;
    uint32_t    seed;
};

struct RadiancePRD
{
    // Quantidade de Luz Emitida
    float3      emitted;
    // Quantidade de Luz Emitida, Transmitida ou Recebida pela Superfície em si
    float3      radiance;
    // Quantidade da Redução da Itensidade da Luz
    float3      attenuation;
    // Origem e Direção da Luz
    float3      origin;
    float3      direction;

    // Float correspondente à Roulette
    float       roulette;

    bool        done;
    uint32_t    seed;
    int32_t     countEmitted;
};

// ############################################################
// Radiance Rays

// Acrescenta-se a ideia de Roleta Russa
// Cria-se uma verificação para as possíveis Texturas dos cenários em Teste
extern "C" __global__ void __closesthit__radiance() 
{
    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    RadiancePRD &prd = *(RadiancePRD *)getPRD<RadiancePRD>();

    // Recuperar Id Primitivo e Índices
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // Obter Barycentric Coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // Compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    const float3 nn = normalize(make_float3(n));

    // Posição da Interceção
    const float3 &rayDir =  optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir ;

    if (prd.countEmitted && length(sbtData.emission) != 0) {
        prd.emitted = sbtData.emission ;
        return;
    }
    else
        prd.emitted = make_float3(0.0f);

    uint32_t seed = prd.seed;

    {
        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        float3 w_in;
        cosine_sample_hemisphere( z1, z2, w_in );
        Onb onb( nn );
        onb.inverse_transform( w_in );
        prd.direction = w_in;
        prd.origin    = pos;

        prd.attenuation *= sbtData.diffuse ;
        prd.countEmitted = false;
    }
    
    // Roleta Russa
    float randomValue = rnd(prd.seed);
    float probability;

    // Caso não se acione a Roulette nas Settings do Composer
    if (optixLaunchParams.global->roulette == 0)
    {
        probability = 1;
    }
    else 
    {
        probability = (prd.attenuation.x + prd.attenuation.y + prd.attenuation.z)/3;
    }

    if(randomValue < probability)
    {
        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        const float3 lightV1 = make_float3(0.47f, 0.0, 0.0f);
        const float3 lightV2 = make_float3(0.0f, 0.0, 0.38f);
        const float3 light_pos = make_float3(optixLaunchParams.global->lightPos) + lightV1 * z1 + lightV2 * z2;

        // Calculate Properties of Light Sample (for area based pdf)
        const float  Ldist = length(light_pos - pos );
        const float3 L     = normalize(light_pos - pos );
        const float  nDl   = dot( nn, L );
        const float3 Ln    = normalize(cross(lightV1, lightV2));
        const float  LnDl  = -dot( Ln, L );

        float weight = 0.0f;
        if( nDl > 0.0f && LnDl > 0.0f )
        {
            uint32_t occluded = 0u;
            optixTrace(optixLaunchParams.traversable,
                pos,
                L,
                0.001f,         // tmin
                Ldist - 0.01f,  // tmax
                0.0f,                    // rayTime
                OptixVisibilityMask( 1 ),
                OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
                SHADOW,      // SBT offset
                RAY_TYPE_COUNT,          // SBT stride
                SHADOW,      // missSBTIndex
                occluded);

            if( !occluded )
            {
                const float att = Ldist * Ldist;
                const float A = length(cross(lightV1, lightV2));
                weight = nDl * LnDl * A  / att;
            }
        }
        
        float3 ldir = make_float3(0.600,-0.400,0.700);
        float i = dot(nn, normalize(-ldir));
        
        // Para renderizar as Texturas no caso de Cenários que as possuam
        if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {
            
            // Compute Pixel Texture Coordinate
            const float4 tc
                = (1.f-u-v) * sbtData.vertexD.texCoord0[index.x]
                +         u * sbtData.vertexD.texCoord0[index.y]
                +         v * sbtData.vertexD.texCoord0[index.z];
            // Fetch Texture Value
            float4 fromTexture = tex2D<float4>(sbtData.texture,tc.x,tc.y);
            
            if (i > 0.8) {
                prd.radiance = make_float3(fromTexture) * i;
            } else {
                prd.radiance = make_float3(fromTexture) * 0.4; 
            }
        }
        else
        {
            prd.radiance += make_float3(5.0f, 5.0f, 5.0f) * weight * optixLaunchParams.global->lightScale/probability;
        }
    }
    else
    {
        prd.done= true;
    }
}

// Esta função não sofre alterações
extern "C" __global__ void __anyhit__radiance()
{
}

// Miss usada para definir a Cor de Fundo
extern "C" __global__ void __miss__radiance()
{
    RadiancePRD &prd = *(RadiancePRD*)getPRD<RadiancePRD>();
    
    // Cor de Fundo representada pelo Float3
    prd.radiance = make_float3(0.0f, 0.0f, 0.0f);
    prd.done = true;
}

//Funções Usadas para o Vidro

extern "C" __global__ void __closesthit__radiance__glass()
{
    const TriangleMeshSBTData &sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
    
    RadiancePRD &prd = *(RadiancePRD*)getPRD<RadiancePRD>();

    // Recuperar Id Primitivo e Índices
    const int primID = optixGetPrimitiveIndex();
    const uint3 index = sbtData.index[primID];

    // Obter Barycentric Coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // Calcular o Vetor Normal à Superfície
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];
    
    if(prd.countEmitted && length(sbtData.emission) != 0)
    {
        prd.emitted = sbtData.emission;
        return;
    }
    else
    {
        prd.emitted = make_float3(0.0f);
    }   

    // Tratar da Parte da Reflexão

    // 1. Normalizar o Vetor Normal
    // O Vetor Normalizado é dado com a mesma direção
    const float3 normalSurface = normalize(make_float3(n));

    // 2. Calcular a Direção do Raio
    const float3 rayDirection = optixGetWorldRayDirection();
    
    // 3. Calcular a Posição
    const float3 position = optixGetWorldRayOrigin() + rayDirection * optixGetRayTmax();
    
    // 4. Aplicar a Reflexão com a Direção do Raio e a Normal à Superfície
    float3 reflection = reflect(rayDirection, normalSurface);

    // Tratar da Parte da Refração
    // Refração vem sempre acompanhada da Reflexão
    // Parte da Luz que incide é Refletida e outra Refratada

    float3 refraction;

    // Produto Escalar do Vetor da Direção do Raio e do Vetor da Normal à Superfície < 0
    if(dot(rayDirection, normalSurface) < 0)
    {
        refraction = refract(rayDirection, normalSurface, 0.6);
    }
    // Produto Escalar do Vetor da Direção do Raio e do Vetor da Normal > 0
    else
    {
        refraction = refract(rayDirection, -normalSurface, 1.50);
    }
    
    prd.attenuation *= sbtData.diffuse;
    prd.countEmitted = true;
    prd.origin = position;
    prd.done = false;

    // Seed é a semente que se usa para gerar o número através da função random(rnd)
    if(rnd(prd.seed) < rnd(prd.seed) * M_PIf)
    {
        prd.direction = rnd(prd.seed) * M_PIf * refraction;
    }
    else
    {
        prd.direction = reflection;
    }
}

//Funções Usadas para o Metal

extern "C" __global__ void __closesthit__radiance__metal()
{
    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    RadiancePRD &prd = *(RadiancePRD *)getPRD<RadiancePRD>();

    // Recuperar Id Primitivo e Índices
    const int primID = optixGetPrimitiveIndex();
    const uint3 index = sbtData.index[primID];

    // Obter Barycentric Coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // Calcular o Vetor Normal à Superfície
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    // Tratar da Parte da Reflexão

    // 1. Normalizar o Vetor Normal
    // O Vetor Normalizado é dado com a mesma direção
    const float3 normalSurface = normalize(make_float3(n));

    // 2. Calcular a Direção do Raio
    const float3 &rayDirection = optixGetWorldRayDirection();
    
    // 3. Calcular a Posição
    const float3 position = optixGetWorldRayOrigin() + rayDirection * optixGetRayTmax();
    
    // 4. Aplicar a Reflexão com a Direção do Raio e a Normal à Superfície
    float3 reflection = reflect(rayDirection, normalSurface);

    if (prd.countEmitted && length(sbtData.emission) != 0)
    {
        prd.emitted = sbtData.emission;
        return;
    }
    else
    {
        prd.emitted = make_float3(0.0f);
    }

    // Pack Pointer essencial para o resultado
    RadiancePRD afterPRD;
    uint32_t u0, u1;
    packPointer(&afterPRD, u0, u1);
    
    // Parte Essencial ao Path Tracing

    // Definir/Determinar o Glossiness (Brilho) e o Glossy Rays (Raios Brilhantes)
    const float glossiness = optixLaunchParams.global->glossiness;
    const int glossyRays = optixLaunchParams.global->glossyRays;
    
    int numberGlossyRays = 0;

    float3 directionLightIncident = make_float3(0.0f);
    float3 glossyMetal = make_float3(0.0f);

    // Lançar o Número de Raios Brilhantes escolhidos aquando da Execução
    while(numberGlossyRays < glossyRays) {

        do{
            const float z1 = rnd(prd.seed);
            const float z2 = rnd(prd.seed);
            
            // Produz o Efeito Glossy ao Metal
            // Hemisphere Sampling consiste em disparar raios em direção ao Hemisférico
            // O w_in no exemplo base passa a ser a Direção da Luz Incidente
            cosine_power_sample_hemisphere(z1, z2, directionLightIncident, glossiness);
            
            Onb onb(reflection);
            onb.inverse_transform(directionLightIncident);
            
            // Atualização dos Valores do RadiancePRD
            prd.direction = directionLightIncident;
            prd.origin = position;
            prd.attenuation *= sbtData.diffuse;

            // Para se poder criar a luz que é refratada da Bola
            prd.countEmitted = true;
        } while (dot(directionLightIncident, normalSurface) < 0);
    
        // Dá o efeito Glossy ao Metal
        // Quanto maior este valor, maior a "bola" de luz que cai sobre o cimo da Superfície
        glossyMetal = glossyMetal + make_float3(0.5f);
        
        ++numberGlossyRays;
    }

    // O restanto do Código Base
    
    const float z1 = rnd(prd.seed);
    const float z2 = rnd(prd.seed);

    const float3 lightV1 = make_float3(0.47f, 0.0, 0.0f);
    const float3 lightV2 = make_float3(0.0f, 0.0, 0.38f);
    const float3 light_pos = make_float3(optixLaunchParams.global->lightPos) + lightV1 * z1 + lightV2 * z2;

    // Calculate Properties of Light Sample
    const float  Ldist = length(light_pos - position);
    const float3 L     = normalize(light_pos - position);
    const float  nDl   = dot(normalSurface, L);
    const float3 Ln    = normalize(cross(lightV1, lightV2));
    const float  LnDl  = -dot(Ln, L);

    float weight = 0.0f;

    if( nDl > 0.0f && LnDl > 0.0f )
    {
        uint32_t occluded = 0u;
        optixTrace(optixLaunchParams.traversable,
            position,
            L,
            0.001f,         // tmin
            Ldist - 0.01f,  // tmax
            0.0f,                    // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_NONE,
            RAIDANCE,      // SBT offset
            RAY_TYPE_COUNT,          // SBT stride
            RAIDANCE,      // missSBTIndex
            u0,u1);

        if(!occluded)
        {
            const float att = Ldist * Ldist;
            const float A = length(cross(lightV1, lightV2));
            weight = nDl * LnDl * A  / att;
        }
    }

    prd.radiance
        += make_float3(5.0f, 5.0f, 5.0f)
        * weight
        * optixLaunchParams.global->lightScale 
        * (glossyMetal/glossyRays);
}

// ############################################################
// Shadow Rays

extern "C" __global__ void __closesthit__shadow()
{
    optixSetPayload_0( static_cast<uint32_t>(true));
}

// Any Hit for Shadows
extern "C" __global__ void __anyhit__shadow()
{
}

// Miss for Shadows
extern "C" __global__ void __miss__shadow()
{
    optixSetPayload_0( static_cast<uint32_t>(false));
}

// ############################################################
// Primary Rays

extern "C" __global__ void __raygen__renderFrame()
{

    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto &camera = optixLaunchParams.camera;  

    const int &maxDepth = optixLaunchParams.frame.maxDepth;

    float squaredRaysPerPixel = float(optixLaunchParams.frame.raysPerPixel);
    float2 delta = make_float2(1.0f/squaredRaysPerPixel, 1.0f/squaredRaysPerPixel);

    float3 result = make_float3(0.0f);

    uint32_t seed = tea<4>( ix * optixGetLaunchDimensions().x + iy, optixLaunchParams.frame.frame );

    for (int i = 0; i < squaredRaysPerPixel; ++i) {
        for (int j = 0; j < squaredRaysPerPixel; ++j) {

            const float2 subpixel_jitter = make_float2( delta.x * (i + rnd(seed)), delta.y * (j + rnd( seed )));
            const float2 screen(make_float2(ix + subpixel_jitter.x, iy + subpixel_jitter.y)
                            / make_float2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y) * 2.0 - 1.0);
        
            // note: nau already takes into account the field of view and ratio when computing 
            // camera horizontal and vertical
            float3 origin = camera.position;
            float3 rayDir = normalize(camera.direction
                                + (screen.x ) * camera.horizontal
                                + (screen.y ) * camera.vertical);

            RadiancePRD prd;
            prd.emitted      = make_float3(0.f);
            prd.radiance     = make_float3(0.f);
            prd.attenuation  = make_float3(1.f);
            prd.countEmitted = true;
            prd.done         = false;
            prd.seed         = seed;

            uint32_t u0, u1;
            packPointer( &prd, u0, u1 );             
            
            for (int k = 0; k < maxDepth && !prd.done; ++k) {

                optixTrace(optixLaunchParams.traversable,
                        origin,
                        rayDir,
                        0.001f,    // tmin
                        1e20f,  // tmax
                        0.0f, OptixVisibilityMask( 1 ),
                        OPTIX_RAY_FLAG_NONE, RAIDANCE, RAY_TYPE_COUNT, RAIDANCE, u0, u1 );

                result += prd.emitted;
                result += prd.radiance * prd.attenuation;

                origin = prd.origin;
                rayDir = prd.direction;
            }
        }
    }

    result = result / (squaredRaysPerPixel*squaredRaysPerPixel);
    float gamma = optixLaunchParams.global->gamma;
    // compute index
    const uint32_t fbIndex = ix + iy*optixGetLaunchDimensions().x;

    optixLaunchParams.global->accumBuffer[fbIndex] = 
        (optixLaunchParams.global->accumBuffer[fbIndex] * optixLaunchParams.frame.subFrame +
        make_float4(result.x, result.y, result.z, 1)) /(optixLaunchParams.frame.subFrame+1);

    
    float4 rgbaf = optixLaunchParams.global->accumBuffer[fbIndex];
    //convert float (0-1) to int (0-255)
    const int r = int(255.0f*min(1.0f, pow(rgbaf.x, 1/gamma)));
    const int g = int(255.0f*min(1.0f, pow(rgbaf.y, 1/gamma)));
    const int b = int(255.0f*min(1.0f, pow(rgbaf.z, 1/gamma))) ;

    // convert to 32-bit rgba value 
    const uint32_t rgba = 0xff000000 | (r<<0) | (g<<8) | (b<<16);
    // write to output buffer
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}