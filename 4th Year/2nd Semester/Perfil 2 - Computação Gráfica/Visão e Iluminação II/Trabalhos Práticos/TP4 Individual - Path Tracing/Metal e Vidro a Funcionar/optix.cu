#include "hip/hip_runtime.h"
 // Our Launch Params
#include "optixParams.h"

extern "C"
{
    __constant__ LaunchParams optixLaunchParams;
}

// Ray Types
enum { RAIDANCE=0, SHADOW, RAY_TYPE_COUNT };

struct ShadowPRD
{
    float       shadowAtt;
    uint32_t    seed;
};

struct RadiancePRD
{
    // Quantidade de Luz Emitida
    float3      emitted;
    // Quantidade de Luz Emitida, Transmitida ou Recebida pela Superfície em si
    float3      radiance;
    // Quantidade da Redução da Itensidade da Luz
    float3      attenuation;
    // Origem e Direção da Luz
    float3      origin;
    float3      direction;

    bool        done;
    uint32_t    seed;
    int32_t     countEmitted;
};

// Esta função não sofre alterações
extern "C" __global__ void __closesthit__radiance() 
{
    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    RadiancePRD &prd = *(RadiancePRD *)getPRD<RadiancePRD>();

    // Recuperar Id Primitivo e Índices
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // Obter Barycentric Coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // Compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    const float3 nn = normalize(make_float3(n));

    // Posição da Interceção
    const float3 &rayDir =  optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir ;

    if (prd.countEmitted && length(sbtData.emission) != 0) {
        prd.emitted = sbtData.emission ;
        return;
    }
    else
        prd.emitted = make_float3(0.0f);

    uint32_t seed = prd.seed;

    {
        const float z1 = rnd(seed);
        const float z2 = rnd(seed);

        float3 w_in;
        cosine_sample_hemisphere( z1, z2, w_in );
        Onb onb( nn );
        onb.inverse_transform( w_in );
        prd.direction = w_in;
        prd.origin    = pos;

        prd.attenuation *= sbtData.diffuse ;
        prd.countEmitted = false;
    }
    
    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd.seed = seed;

    const float3 lightV1 = make_float3(0.47f, 0.0, 0.0f);
    const float3 lightV2 = make_float3(0.0f, 0.0, 0.38f);
    const float3 light_pos = make_float3(optixLaunchParams.global->lightPos) + lightV1 * z1 + lightV2 * z2;

    // Calculate properties of Light Sample (for area based pdf)
    const float  Ldist = length(light_pos - pos );
    const float3 L     = normalize(light_pos - pos );
    const float  nDl   = dot( nn, L );
    const float3 Ln    = normalize(cross(lightV1, lightV2));
    const float  LnDl  = -dot( Ln, L );

    float weight = 0.0f;
    if( nDl > 0.0f && LnDl > 0.0f )
    {
        uint32_t occluded = 0u;
        optixTrace(optixLaunchParams.traversable,
            pos,
            L,
            0.001f,         // tmin
            Ldist - 0.01f,  // tmax
            0.0f,                    // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
            SHADOW,      // SBT offset
            RAY_TYPE_COUNT,          // SBT stride
            SHADOW,      // missSBTIndex
            occluded);

        if( !occluded )
        {
            const float att = Ldist * Ldist;
            const float A = length(cross(lightV1, lightV2));
            weight = nDl * LnDl * A  / att;
        }
    }

    prd.radiance += make_float3(5.0f, 5.0f, 5.0f) * weight * optixLaunchParams.global->lightScale;
}

// Esta função não sofre alterações
extern "C" __global__ void __anyhit__radiance()
{
}

// Miss usada para definir a Cor de Fundo
// Cor mudada de float3(0,0,0) para float3(1,1,1)
// Fundo branco. Nota-se no Reflexo do Objecto.
extern "C" __global__ void __miss__radiance()
{
    RadiancePRD &prd = *(RadiancePRD*)getPRD<RadiancePRD>();
    // Cor de Fundo representada pelo Float3
    prd.radiance = make_float3(1.0f, 1.0f, 1.0f);
    prd.done = true;
}

// 1. Shadow Rays

extern "C" __global__ void __closesthit__shadow()
{
    optixSetPayload_0( static_cast<uint32_t>(true));
}

// any hit for shadows
extern "C" __global__ void __anyhit__shadow()
{
}

// miss for shadows
extern "C" __global__ void __miss__shadow()
{
    optixSetPayload_0( static_cast<uint32_t>(false));
}

//METAL

extern "C" __global__ void __closesthit__radiance_metal()
{

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  


    RadiancePRD &prd = *(RadiancePRD *)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    const float3 normal = normalize(make_float3(n));
    float3 normRayDir = optixGetWorldRayDirection();

    // intersection position
    const float3 &rayDir =  optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir;

    if (prd.countEmitted && length(sbtData.emission) != 0) {
        prd.emitted = sbtData.emission;
        return;
    }
    else
        prd.emitted = make_float3(0.0f);

    //declaração do seed  
    uint32_t seed = prd.seed;
    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd.seed = seed;

    RadiancePRD afterPRD;
    afterPRD.radiance = make_float3(1.0f);
    afterPRD.seed = prd.seed;
    uint32_t u0, u1;
    packPointer( &afterPRD, u0, u1 );
    
    //Achar o número de raios brilhantes, e o grau de brilho
    const float glossiness = optixLaunchParams.global->glossiness;
    const int glossy_rays = optixLaunchParams.global->glossyRays;
    float3 glossy = make_float3(0.0f);
    float3 direcaoL;
    float3 reflectDir = reflect(optixGetWorldRayDirection(), normal);
    
    //Lançar os raios
    for (int i = 0; i < glossy_rays; ++i) {
        afterPRD.radiance = make_float3(1.0f);
        do{
            const float z1 = rnd(seed);
            const float z2 = rnd(seed);
            prd.seed = seed;
        
            cosine_power_sample_hemisphere( z1, z2, direcaoL, glossiness );
            Onb onb( reflectDir );
            onb.inverse_transform( direcaoL );
            prd.direction = direcaoL;
            prd.origin    = pos;
            prd.attenuation *= sbtData.diffuse ;
           // prd.countEmitted = false;
        } while (dot(direcaoL, normal) < 0.001);
    
        glossy += afterPRD.radiance;
    
    }

    const float3 lightV1 = make_float3(0.47f, 0.0, 0.0f);
    const float3 lightV2 = make_float3(0.0f, 0.0, 0.38f);
    const float3 light_pos = make_float3(optixLaunchParams.global->lightPos) + lightV1 * z1 + lightV2 * z2;

    // Calculate properties of light sample (for area based pdf)
    const float  Ldist = length(light_pos - pos );
    const float3 L     = normalize(light_pos - pos );
    const float  nDl   = dot( normal, L );
    const float3 Ln    = normalize(cross(lightV1, lightV2));
    const float  LnDl  = -dot( Ln, L );

    float weight = 0.0f;
    if( nDl > 0.0f && LnDl > 0.0f )
    {
    uint32_t occluded = 0u;
        optixTrace(optixLaunchParams.traversable,
            pos,
            L,
            0.001f,         // tmin
            Ldist - 0.01f,  // tmax
            0.0f,                    // rayTime
            OptixVisibilityMask( 1 ),
            OPTIX_RAY_FLAG_NONE,
            RAIDANCE,      // SBT offset
            RAY_TYPE_COUNT,          // SBT stride
            RAIDANCE,      // missSBTIndex
            u0,u1);

    if( !occluded )
        {
            const float att = Ldist * Ldist;
            const float A = length(cross(lightV1, lightV2));
        weight = nDl * LnDl * A  / att;
        }
    }
    prd.seed = seed;
    prd.radiance += make_float3(5.0f, 5.0f, 5.0f)*weight* optixLaunchParams.global->lightScale * (glossy *make_float3(0.8f, 0.8f, 0.8f) / glossy_rays);
}

//GLASS

extern "C" __global__ void __closesthit__radiance_glass()
{
    const TriangleMeshSBTData &sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
    
    RadiancePRD &prd = *(RadiancePRD*)getPRD<RadiancePRD>();

    // retrieve primitive id and indexes
    const int primID = optixGetPrimitiveIndex();
    const uint3 index = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];
    
    float3 normal = normalize(make_float3(n));
    const float3 normRayDir = optixGetWorldRayDirection();

    // intersection position
    const float3 &rayDir = optixGetWorldRayDirection();
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * rayDir;
    float3 reflectDir = reflect(normRayDir, normal);

    if(prd.countEmitted && length(sbtData.emission) != 0)
    {
        prd.emitted = sbtData.emission;
        return;
    }
    else
    {
        prd.emitted = make_float3(0.0f);
    }

    uint32_t seed = prd.seed;

    const float z1 = rnd(seed);
    const float z2 = rnd(seed);
    prd.seed = seed;
    const float z = rnd(prd.seed);

    const float3 lightV1 = make_float3(0.47f, 0.0, 0.0f);
    const float3 lightV2 = make_float3(0.0f, 0.0, 0.38f);
    const float3 light_pos = make_float3(optixLaunchParams.global->lightPos) + lightV1 * z1 + lightV2 * z2;

    // calculate properties of light sample (for area based pdf)
    const float Ldist = length(light_pos - pos);
    const float3 L = normalize(light_pos - pos);
    const float nDl = dot(normal,L);
    const float3 Ln = normalize(cross(lightV1, lightV2));
    const float LnDl = -dot(Ln, L);

    // nova direcao do raio
    float3 direcao;

    // entering glass
    float dotP;
    if(dot(normRayDir, normal) < 0)
    {
        dotP = dot(normRayDir, -normal);

        direcao = refract(normRayDir, normal, 0.66);
        prd.direction = direcao;
        prd.attenuation *= sbtData.diffuse;
        prd.countEmitted = true;
        prd.origin = pos;
        prd.done = false;
        prd.seed = seed; 
    }
    // exiting glass
    else
    {
        dotP = 0;
        
        direcao = refract(normRayDir, -normal, 1.5);
        prd.direction = direcao;
        prd.attenuation *= sbtData.diffuse;
        prd.countEmitted = true;
        prd.origin = pos;
        prd.done = false;
        prd.seed = seed; 
    }

    RadiancePRD refractPRD;
    refractPRD.radiance = make_float3(0.0f);
    refractPRD.seed = prd.seed;
    uint32_t u0, u1;
    packPointer(&refractPRD, u0, u1);

    float weight = 0.0f;
    if(nDl > 0.0f && LnDl > 0.0f)
    {
        uint32_t occluded = 0u;
        if(length(direcao) > 0)
        {
            optixTrace(optixLaunchParams.traversable,
                pos,
                direcao,
                0.00001f,
                Ldist - 0.01f,
                0.0f,
                OptixVisibilityMask(1),
                OPTIX_RAY_FLAG_NONE,
                RAIDANCE,
                RAY_TYPE_COUNT,
                RAIDANCE,
                u0,u1);
        }
        RadiancePRD reflectPRD;
        reflectPRD.radiance = make_float3(0.0f);
        reflectPRD.seed = prd.seed;

        if(dotP > 0)
        {
            packPointer(&reflectPRD, u0, u1);

            //Achar o número de raios brilhantes, e o grau de brilho
            const float glossiness = optixLaunchParams.global->glossiness;
            const int glossy_rays = optixLaunchParams.global->glossyRays;
            float3 glossy = make_float3(0.0f);
            float3 direcaoL;
            float3 reflectDir = reflect(optixGetWorldRayDirection(), normal);
            
            //Lançar os raios
            for (int i = 0; i < glossy_rays; ++i) {
                reflectPRD.radiance = make_float3(1.0f);
                do{
                    const float z1 = rnd(seed);
                    const float z2 = rnd(seed);
                    prd.seed = seed;
                
                    cosine_power_sample_hemisphere( z1, z2, direcaoL, glossiness );
                    Onb onb( reflectDir );
                    onb.inverse_transform( direcaoL );
                    prd.direction = direcaoL;
                    prd.origin    = pos;
                    prd.attenuation *= sbtData.diffuse ;
                // prd.countEmitted = false;
                } while (dot(direcaoL, normal) < 0.001);
            
                glossy += reflectPRD.radiance;
            
            }

            optixTrace(optixLaunchParams.traversable,
                pos,
                L,
                0.00001f,
                Ldist - 0.01f,
                0.0f,
                OptixVisibilityMask(1),
                OPTIX_RAY_FLAG_NONE,
                RAIDANCE,
                RAY_TYPE_COUNT,
                RAIDANCE,
                u0,u1);
            
            float r0 = (15.0f - 1.0f)/(15.0f + 1.0f);
            r0 = r0*r0 + (1-r0*r0) * pow(1-dotP, 5);
            prd.radiance += make_float3(5.0f, 5.0f, 5.0f)*(weight)*optixLaunchParams.global->lightScale * (refractPRD.radiance * (1-r0) + r0 * reflectPRD.radiance);
        }
        else
        {
            prd.radiance += make_float3(5.0f, 5.0f, 5.0f)*(weight)*optixLaunchParams.global->lightScale * refractPRD.radiance;
        }
        if(!occluded)
        {
            const float att = Ldist * Ldist;
            const float A = length(cross(lightV1,lightV2));
            weight = nDl * LnDl * A / att;
        }
    }

    if(rnd(prd.seed) > 0.5)
    {
        prd.direction = direcao;
    }
    else
    {
        prd.direction = reflectDir;
    }

}

extern "C" __global__ void __anyhit__radiance_glass()
{
}

// miss sets the background color
extern "C" __global__ void __miss__radiance_glass()
{
    RadiancePRD &prd = *(RadiancePRD*)getPRD<RadiancePRD>();
    // set black as background color
    prd.radiance = make_float3(0.0f, 0.0f, 0.0f);
    prd.done = true;
}

// Shadow rays

extern "C" __global__ void __closesthit__shadow_glass()
{
    optixSetPayload_0( static_cast<uint32_t>(true));
}

// any hit for shadows
extern "C" __global__ void __anyhit__shadow_glass()
{
}

// miss for shadows
extern "C" __global__ void __miss__shadow_glass()
{
    optixSetPayload_0( static_cast<uint32_t>(false));
}

// -----------------------------------------------
// Primary Rays


extern "C" __global__ void __raygen__renderFrame()
{

    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto &camera = optixLaunchParams.camera;  

    const int &maxDepth = optixLaunchParams.frame.maxDepth;
 
    float squaredRaysPerPixel = float(optixLaunchParams.frame.raysPerPixel);
    float2 delta = make_float2(1.0f/squaredRaysPerPixel, 1.0f/squaredRaysPerPixel);

    float3 result = make_float3(0.0f);

    uint32_t seed = tea<4>( ix * optixGetLaunchDimensions().x + iy, optixLaunchParams.frame.frame );

    for (int i = 0; i < squaredRaysPerPixel; ++i) {
        for (int j = 0; j < squaredRaysPerPixel; ++j) {

            const float2 subpixel_jitter = make_float2( delta.x * (i + rnd(seed)), delta.y * (j + rnd( seed )));
            const float2 screen(make_float2(ix + subpixel_jitter.x, iy + subpixel_jitter.y)
                            / make_float2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y) * 2.0 - 1.0);
        
            // note: nau already takes into account the field of view and ratio when computing 
            // camera horizontal and vertical
            float3 origin = camera.position;
            float3 rayDir = normalize(camera.direction
                                + (screen.x ) * camera.horizontal
                                + (screen.y ) * camera.vertical);

            RadiancePRD prd;
            prd.emitted      = make_float3(0.f);
            prd.radiance     = make_float3(0.f);
            prd.attenuation  = make_float3(1.f);
            prd.countEmitted = true;
            prd.done         = false;
            prd.seed         = seed;

            uint32_t u0, u1;
            packPointer( &prd, u0, u1 );             
            
            for (int k = 0; k < maxDepth && !prd.done; ++k) {

                optixTrace(optixLaunchParams.traversable,
                        origin,
                        rayDir,
                        0.001f,    // tmin
                        1e20f,  // tmax
                        0.0f, OptixVisibilityMask( 1 ),
                        OPTIX_RAY_FLAG_NONE, RAIDANCE, RAY_TYPE_COUNT, RAIDANCE, u0, u1 );

                result += prd.emitted;
                result += prd.radiance * prd.attenuation;

                origin = prd.origin;
                rayDir = prd.direction;
            }
        }
    }

    result = result / (squaredRaysPerPixel*squaredRaysPerPixel);
    float gamma = optixLaunchParams.global->gamma;
    // compute index
    const uint32_t fbIndex = ix + iy*optixGetLaunchDimensions().x;

    optixLaunchParams.global->accumBuffer[fbIndex] = 
        (optixLaunchParams.global->accumBuffer[fbIndex] * optixLaunchParams.frame.subFrame +
        make_float4(result.x, result.y, result.z, 1)) /(optixLaunchParams.frame.subFrame+1);

    
    float4 rgbaf = optixLaunchParams.global->accumBuffer[fbIndex];
    //convert float (0-1) to int (0-255)
    const int r = int(255.0f*min(1.0f, pow(rgbaf.x, 1/gamma)));
    const int g = int(255.0f*min(1.0f, pow(rgbaf.y, 1/gamma)));
    const int b = int(255.0f*min(1.0f, pow(rgbaf.z, 1/gamma))) ;

    // convert to 32-bit rgba value 
    const uint32_t rgba = 0xff000000 | (r<<0) | (g<<8) | (b<<16);
    // write to output buffer
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}