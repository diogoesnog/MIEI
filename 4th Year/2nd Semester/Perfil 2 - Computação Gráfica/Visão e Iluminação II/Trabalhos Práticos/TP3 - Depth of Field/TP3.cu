#include "hip/hip_runtime.h"

#include <optix.h>
#include "random.h"
#include "LaunchParams7.h" // our launch params
#include <vec_math.h> // NVIDIAs math utils


extern "C" {
    __constant__ LaunchParams optixLaunchParams;
}
//  a single ray type
enum { PHONG=0, SHADOW, RAY_TYPE_COUNT };


// -------------------------------------------------------
// closest hit computes color based lolely on the triangle normal

extern "C" __global__ void __closesthit__radiance() {

    float3 &prd = *(float3*)getPRD<float3>();

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    // intersection position
    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();

    // direction towards light
    float3 lPos = make_float3(optixLaunchParams.global->lightPos);
    float lDirLength = length(lPos - pos) - 0.01f;
    float3 lDir = normalize(lPos - pos);
    float3 nn = normalize(make_float3(n));

    float intensity = max(dot(lDir, nn),0.0f);

    // ray payload
    float shadowAttPRD = 1.0f;
    uint32_t u0, u1;
    packPointer( &shadowAttPRD, u0, u1 );  
  
    // trace shadow ray
    optixTrace(optixLaunchParams.traversable,
        pos,
        lDir,
        0.001f,         // tmin
        lDirLength,     // tmax
        0.0f,           // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
        SHADOW,             // SBT offset
        RAY_TYPE_COUNT,     // SBT stride
        SHADOW,             // missSBTIndex 
        u0, u1 );

    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {  
        // get barycentric coordinates
        // compute pixel texture coordinate
        const float4 tc
          = (1.f-u-v) * sbtData.vertexD.texCoord0[index.x]
          +         u * sbtData.vertexD.texCoord0[index.y]
          +         v * sbtData.vertexD.texCoord0[index.z];
        // fetch texture value
        
        float4 fromTexture = tex2D<float4>(sbtData.texture,tc.x,tc.y);
        prd = make_float3(fromTexture) * min(intensity * shadowAttPRD + 0.0, 1.0);
    }
    else
        prd = sbtData.color * min(intensity * shadowAttPRD + 0.0, 1.0);
}


// any hit to ignore intersections with back facing geometry
extern "C" __global__ void __anyhit__radiance() {

}


// miss sets the background color
extern "C" __global__ void __miss__radiance() {

    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(0.0f, 0.0f, 1.0f);
}


// -----------------------------------------------
// Shadow rays

extern "C" __global__ void __closesthit__shadow() {

    float &prd = *(float*)getPRD<float>();
    prd = 0.0f;
}


// any hit for shadows
extern "C" __global__ void __anyhit__shadow() {

}


// miss for shadows
extern "C" __global__ void __miss__shadow() {

    float &prd = *(float*)getPRD<float>();
    // set blue as background color
    prd = 1.0f;
}


// -----------------------------------------------
// Light material


extern "C" __global__ void __closesthit__light() {

    float3 &prd = *(float3*)getPRD<float3>();
    prd = make_float3(1.0f, 1.0f, 1.0f);
}


extern "C" __global__ void __anyhit__light() {
}


extern "C" __global__ void __miss__light() {
}


extern "C" __global__ void __closesthit__light_shadow() {

    float &prd = *(float*)getPRD<float>();
    prd = 1.0f;
}


// any hit to ignore intersections based on alpha transparency
extern "C" __global__ void __anyhit__light_shadow() {
}


// miss sets the background color
extern "C" __global__ void __miss__light_shadow() {
}




// -----------------------------------------------
// Metal Phong rays

extern "C" __global__ void __closesthit__phong_metal() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];
    // ray payload

    float3 normal = normalize(make_float3(n));

    // entering glass
    //if (dot(optixGetWorldRayDirection(), normal) < 0)

    float3 afterPRD = make_float3(1.0f);
    uint32_t u0, u1;
    packPointer( &afterPRD, u0, u1 );  

    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();
    //(1.f-u-v) * A + u * B + v * C;
    
    float3 rayDir = reflect(optixGetWorldRayDirection(), normal);
    optixTrace(optixLaunchParams.traversable,
        pos,
        rayDir,
        0.00001f,    // tmin
        1e20f,  // tmax
        0.0f,   // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
        PHONG,             // SBT offset
        RAY_TYPE_COUNT,     // SBT stride
        PHONG,             // missSBTIndex 
        u0, u1 );

    float3 &prd = *(float3*)getPRD<float3>();
    prd = make_float3(0.8,0.8,0.8) * afterPRD;
}





// -----------------------------------------------
// Glass Phong rays

SUTIL_INLINE SUTIL_HOSTDEVICE float3 refract(const float3& i, const float3& n, const float eta) {

    float k = 1.0 - eta * eta * (1.0 - dot(n, i) * dot(n, i));
    if (k < 0.0)
        return make_float3(0.0f);
    else
        return (eta * i - (eta * dot(n, i) + sqrt(k)) * n);
}


extern "C" __global__ void __closesthit__phong_glass() {

    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    // compute normal
    const float4 n
        = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        +         u * sbtData.vertexD.normal[index.y]
        +         v * sbtData.vertexD.normal[index.z];

    float3 normal = normalize(make_float3(n));
    const float3 normRayDir = optixGetWorldRayDirection();

    // new ray direction
    float3 rayDir;
    // entering glass
    float dotP;
    if (dot(normRayDir, normal) < 0) {
        dotP = dot(normRayDir, -normal);
        rayDir = refract(normRayDir, normal, 0.66);
    }
    // exiting glass
    else {
        dotP = 0;
        rayDir = refract(normRayDir, -normal, 1.5);
    }

    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax() * optixGetWorldRayDirection();
    
    float3 refractPRD = make_float3(0.0f);
    uint32_t u0, u1;
    packPointer( &refractPRD, u0, u1 );  
    
    if (length(rayDir) > 0)
        optixTrace(optixLaunchParams.traversable,
            pos,
            rayDir,
            0.00001f,    // tmin
            1e20f,  // tmax
            0.0f,   // rayTime
            OptixVisibilityMask( 255 ),
            OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
            PHONG,             // SBT offset
            RAY_TYPE_COUNT,     // SBT stride
            PHONG,             // missSBTIndex 
            u0, u1 );

    // ray payload 
    float3 &prd = *(float3*)getPRD<float3>();
 
    float3 reflectPRD = make_float3(0.0f);
    if (dotP > 0) {
        float3 reflectDir = reflect(normRayDir, normal);        
        packPointer( &reflectPRD, u0, u1 );  
        optixTrace(optixLaunchParams.traversable,
            pos,
            reflectDir,
            0.00001f,    // tmin
            1e20f,  // tmax
            0.0f,   // rayTime
            OptixVisibilityMask( 255 ),
            OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
            PHONG,             // SBT offset
            RAY_TYPE_COUNT,     // SBT stride
            PHONG,             // missSBTIndex 
            u0, u1 );
        float r0 = (1.5f - 1.0f)/(1.5f + 1.0f);
        r0 = r0*r0 + (1-r0*r0) * pow(1-dotP,5);
        prd =  refractPRD * (1-r0) + r0*reflectPRD;
    }
    else
        prd =  refractPRD ;
}



extern "C" __global__ void __anyhit__phong_glass() {

}


// miss sets the background color
extern "C" __global__ void __miss__phong_glass() {

    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(0.0f, 0.0f, 1.0f);
}



// -----------------------------------------------
// Glass Shadow rays

extern "C" __global__ void __closesthit__shadow_glass() {

    // ray payload
    float afterPRD = 1.0f;
    uint32_t u0, u1;
    packPointer( &afterPRD, u0, u1 );  

    const float3 pos = optixGetWorldRayOrigin() + optixGetRayTmax()*optixGetWorldRayDirection();
    
    // trace primary ray
    optixTrace(optixLaunchParams.traversable,
        pos,
        optixGetWorldRayDirection(),
        0.001f,    // tmin
        1e20f,  // tmax
        0.0f,   // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_NONE, //OPTIX_RAY_FLAG_NONE,
        SHADOW,             // SBT offset
        RAY_TYPE_COUNT,     // SBT stride
        SHADOW,             // missSBTIndex 
        u0, u1 );

    float &prd = *(float*)getPRD<float>();
    prd = 0.95f * afterPRD;
}


// any hit for shadows
extern "C" __global__ void __anyhit__shadow_glass() {

}


// miss for shadows
extern "C" __global__ void __miss__shadow_glass() {

    float &prd = *(float*)getPRD<float>();
    // set blue as background color
    prd = 1.0f;
}


// -----------------------------------------------
// Primary Rays

extern "C" __global__ void __raygen__renderFrame() {

    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto &camera = optixLaunchParams.camera;  
    
    float raysPerPixel = float(optixLaunchParams.frame.raysPerPixel);
    float2 delta = make_float2(1.0f/raysPerPixel, 1.0f/raysPerPixel);

    const float d_o = optixLaunchParams.global->focalDistance;
    const float A = optixLaunchParams.global->aperture;
    const float d_p = optixLaunchParams.global->lensDistance;
 
	if (optixLaunchParams.frame.frame == 0 && ix == 0 && iy == 0) {

		// print info to console
		printf("===========================================\n");
        printf("Nau Ray-Tracing Debug\n");
        const float4 &ld = optixLaunchParams.global->lightPos;
        printf("LightPos: %f, %f %f %f\n", ld.x,ld.y,ld.z,ld.w);
        printf("Launch dim: %u %u\n", optixGetLaunchDimensions().x, optixGetLaunchDimensions().y);
        printf("===========================================\n");
	}

    // ray payload
    float3 pixelColorPRD = make_float3(1.f);
    uint32_t u0, u1;
    packPointer( &pixelColorPRD, u0, u1 );

    // compute ray direction
    // normalized screen plane position, in [-1, 1]^2
    const float2 screen(make_float2(ix+.5f,iy+.5f)
                    / make_float2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y) * 2.0 - 1.0);


    // TP3

    //Cálculo coordenadas do pixel
    float3 pixel = camera.position + (screen.x * (-1)) * camera.horizontal + (screen.y * (-1)) * camera.vertical;

    //Cálculo direção do pixel à camara
    float3 dir_pixel_camera = pixel - camera.position;

    //Cálculo do ângulo entre o vetor do pixel ao centro da camera e do vetor do centro da camera ao centro do Plano de Imagem
    float angle_dcp_dcpi = acos((dir_pixel_camera.x * camera.direction.x + dir_pixel_camera.y * camera.direction.y + dir_pixel_camera.z * camera.direction.z)/(sqrt (pow(dir_pixel_camera.x, 2) + pow(dir_pixel_camera.y, 2) + pow(dir_pixel_camera.z, 2)) * sqrt((pow(camera.direction.x, 2) + pow(camera.direction.y, 2) + pow(camera.direction.z, 2)))));

    //Cálculo da distância entre o pixel ao centro da camera
    float dpc = sqrt(pow(pixel.x - camera.position.x, 2) + pow(pixel.y - camera.position.y, 2) + pow(pixel.z - camera.position.z, 2)); 
    
    //Cálculo da distância entre o centro da camera e o centro do Plano de Imagem
    float dc_planoImagem = cos(angle_dcp_dcpi) * dpc;

    //Cálculo das coordenadas do centro da lente
    float3 centroLente = camera.position + camera.direction * (d_p - dc_planoImagem);

    // Cálculo da distância do pixel ao centro da lente
    float dpl = sqrt(pow(pixel.x - centroLente.x, 2) + pow(pixel.y - centroLente.y, 2) + pow(pixel.z - centroLente.z, 2)); 


    //Cálculo do ponto P (PONTO DO PLANO DE FOCO)
    float m = d_o / d_p; // factor de proporção

    float dlP = dpl * m; // Distância do centro da lente ao ponto P 

    float3 P = pixel + normalize(centroLente - pixel) * (dpl + dlP); // Coordenadas do P no plano de foco
    


    float red = 0.0f, blue = 0.0f, green = 0.0f;
    for (int i = 0; i < raysPerPixel; ++i) {
        for (int j = 0; j < raysPerPixel; ++j) {
            uint32_t seed = tea<4>( ix * optixGetLaunchDimensions().x + iy, i*raysPerPixel + j );

            float raio = A * rnd(seed);
            float angulo = 2 * M_PIf * rnd(seed);

            float3 frente = normalize(camera.direction);

            float dot_perp = dot(frente, make_float3(1,0,0));

            float3 perpendicular;

            if (dot_perp < 0.1) {
                perpendicular = normalize(cross(make_float3(1,0,0), frente));
            } else {
                perpendicular = normalize(cross(make_float3(0,1,0), frente));
            }

            float3 outra_perp = normalize(cross(perpendicular, frente));

            //Cálculo do ponto de origem para lançar o raio
            float3 ponto_sampling = centroLente + raio * (perpendicular * cos (angulo) + outra_perp * sin(angulo));
            
            //Cálculo da direção dos raios
            float3 direcao = normalize(P - ponto_sampling);
            
            // trace primary ray
            optixTrace(optixLaunchParams.traversable,
                    ponto_sampling,
                    direcao,
                    0.f,    // tmin
                    1e20f,  // tmax
                    0.0f,   // rayTime
                    OptixVisibilityMask( 255 ),
                    OPTIX_RAY_FLAG_NONE,//,OPTIX_RAY_FLAG_DISABLE_ANYHIT
                    PHONG,             // SBT offset
                    RAY_TYPE_COUNT,               // SBT stride
                    PHONG,             // missSBTIndex 
                    u0, u1 );

            red += pixelColorPRD.x / (raysPerPixel*raysPerPixel);
            green += pixelColorPRD.y / (raysPerPixel*raysPerPixel);
            blue += pixelColorPRD.z / (raysPerPixel*raysPerPixel);

            if (ix == 0 && iy == 0) {
                printf("pixel: %f %f %f\n", pixel.x, pixel.y, pixel.z);
                printf("centro camera: %f %f %f\n", camera.position.x, camera.position.y, camera.position.z);
            }
        }
    }

    //convert float (0-1) to int (0-255)
    const int r = int(255.0f*red);
    const int g = int(255.0f*green);
    const int b = int(255.0f*blue);
    // convert to 32-bit rgba value 
    const uint32_t rgba = 0xff000000
      | (r<<0) | (g<<8) | (b<<16);
    // compute index
    const uint32_t fbIndex = ix + iy*optixGetLaunchDimensions().x;
    // write to output buffer
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}
  

