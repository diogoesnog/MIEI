#include "hip/hip_runtime.h"
#include <optix.h>
#include "LaunchParams.h" // our launch params
#include "vec_math.h" // NVIDIAs math utils

extern "C" {
    __constant__ LaunchParams optixLaunchParams;
}

// Acrescentamos o SHADOW_RAY_TYPE
enum { PHONG_RAY_TYPE=0, SHADOW_RAY_TYPE, RAY_TYPE_COUNT};

extern "C" __global__ void __closesthit__radiance() {
    // get the payload variable
    float3 &prd = *(float3*)getPRD<float3>();

    // get mesh data
    const TriangleMeshSBTData &sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // ray payload
    float3 pixelColorPRD = make_float3(1.f);
    uint32_t u0, u1;
    packPointer( &pixelColorPRD, u0, u1 );  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    float4 pos4 = (1.f-u-v) * sbtData.vertexD.position[index.x]
        + u * sbtData.vertexD.position[index.y]
        + v * sbtData.vertexD.position[index.z];

    float3 pos = make_float3(pos4.x,pos4.y,pos4.z);

    float4 norm4 = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        + u * sbtData.vertexD.normal[index.y]
        + v * sbtData.vertexD.normal[index.z];

    float3 normal = make_float3(norm4.x,norm4.y,norm4.z);

    float3 ldir = make_float3(0.600,-0.400,0.700);

    float i = dot(normalize(normal), normalize(-ldir));

    optixTrace(optixLaunchParams.traversable,
            pos,
            -ldir,
            0.1f, // tmin
            1e20f, // tmax
            0.0f, // rayTime
            OptixVisibilityMask( 255 ),
            OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
            SHADOW_RAY_TYPE, // SBT offset
            RAY_TYPE_COUNT, // SBT stride
            SHADOW_RAY_TYPE, // missSBTIndex
            u0, u1 );

    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {
        // compute pixel texture coordinate
        const float4 tc
            = (1.f-u-v) * sbtData.vertexD.texCoord0[index.x]
            +         u * sbtData.vertexD.texCoord0[index.y]
            +         v * sbtData.vertexD.texCoord0[index.z];
        // fetch texture value
        float4 fromTexture = tex2D<float4>(sbtData.texture,tc.x,tc.y);
        
        if (i > 0.8) {
            prd= make_float3(fromTexture) * i;
        } else {
            prd= make_float3(fromTexture) * 0.4; 
        }
    }
    else {
        if (i > 0.3) {
            prd = sbtData.color * i;
        } else {
            prd = sbtData.color * 0.3;
        }
    }
    if(pixelColorPRD.y == 1.0) {
        prd *= 0.5;
    }
}

// Nothing to do in here
extern "C" __global__ void __anyhit__radiance() {}

// Miss sets the Background Color
extern "C" __global__ void __miss__radiance() {

    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(1.0f, 1.0f, 1.0f);
}

extern "C" __global__ void __closesthit__shadow(){
    // get the payload variable
    float3 &prd = *(float3*)getPRD<float3>();
    prd=make_float3(0,1,0);
}

// Nothing to do in here
extern "C" __global__ void __anyhit__shadow() {}

extern "C" __global__ void __miss__shadow() {
    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(1.0f, 0.0f, 0.0f);
}

// GRADES PHON FUNCTIONS

extern "C" __global__ void __closesthit__phong_alphaTrans() {}

extern "C" __global__ void __anyhit__phong_alphaTrans() {
    // get the payload variable
    float3 &prd = *(float3*)getPRD<float3>();

    // get mesh data
    const TriangleMeshSBTData &sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // ray payload
    float3 pixelColorPRD = make_float3(1.f);
    uint32_t u0, u1;
    packPointer( &pixelColorPRD, u0, u1 );  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    float4 norm4 = (1.f-u-v) * sbtData.vertexD.normal[index.x]
    + u * sbtData.vertexD.normal[index.y]
    + v * sbtData.vertexD.normal[index.z];

    float3 normal = make_float3(norm4.x,norm4.y,norm4.z);

    float3 ldir = make_float3(0.600,-0.400,0.700);

    float i = dot(normalize(normal), normalize(-ldir));

    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {
        // compute pixel texture coordinate
        const float4 tc
            = (1.f-u-v) * sbtData.vertexD.texCoord0[index.x]
            +         u * sbtData.vertexD.texCoord0[index.y]
            +         v * sbtData.vertexD.texCoord0[index.z];
        // fetch texture value
        float4 fromTexture = tex2D<float4>(sbtData.texture,tc.x,tc.y);

        if (fromTexture.w == 1) {
            if (i > 0.3) {
                prd= make_float3(fromTexture) * i;
            } else {
                prd= make_float3(fromTexture) * 0.3; 
            }
        }
        else {
            optixIgnoreIntersection();
        }
    }
}

extern "C" __global__ void __miss__phong_alphaTrans() {
}

// GRADES SHADOW FUNCTIONS

extern "C" __global__ void __closesthit__shadow_alphaTrans() {}

extern "C" __global__ void __anyhit__shadow_alphaTrans() {
    // get the payload variable
    float3 &prd = *(float3*)getPRD<float3>();

    // get mesh data
    const TriangleMeshSBTData &sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // ray payload
    float3 pixelColorPRD = make_float3(1.f);
    uint32_t u0, u1;
    packPointer( &pixelColorPRD, u0, u1 );  

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    float4 norm4 = (1.f-u-v) * sbtData.vertexD.normal[index.x]
    + u * sbtData.vertexD.normal[index.y]
    + v * sbtData.vertexD.normal[index.z];

    float3 normal = make_float3(norm4.x,norm4.y,norm4.z);

    float3 ldir = make_float3(0.600,-0.400,0.700);

    float i = dot(normalize(normal), normalize(-ldir));

    if (sbtData.hasTexture && sbtData.vertexD.texCoord0) {
        // compute pixel texture coordinate
        const float4 tc
            = (1.f-u-v) * sbtData.vertexD.texCoord0[index.x]
            +         u * sbtData.vertexD.texCoord0[index.y]
            +         v * sbtData.vertexD.texCoord0[index.z];
        // fetch texture value
        float4 fromTexture = tex2D<float4>(sbtData.texture,tc.x,tc.y);

        if (fromTexture.w != 1) {
            optixIgnoreIntersection();
        }
    }
}

extern "C" __global__ void __miss__shadow_alphaTrans() {
    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(1.0f, 0.0f, 0.0f);
}

// GLASS FUNCTIONS

// Nothing to do in here
extern "C" __global__ void __anyhit__phong_glass() {}

extern "C" __global__ void __closesthit__phong_glass() {
    // get the payload variable
    float3 &prd = *(float3*)getPRD<float3>();

    // get mesh data
    const TriangleMeshSBTData &sbtData
        = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // retrieve primitive id and indexes
    const int   primID = optixGetPrimitiveIndex();
    const uint3 index  = sbtData.index[primID];

    // get barycentric coordinates
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    float4 pos4 = (1.f-u-v) * sbtData.vertexD.position[index.x]
        + u * sbtData.vertexD.position[index.y]
        + v * sbtData.vertexD.position[index.z];

    float3 pos = make_float3(pos4.x,pos4.y,pos4.z);

    float4 norm4 = (1.f-u-v) * sbtData.vertexD.normal[index.x]
        + u * sbtData.vertexD.normal[index.y]
        + v * sbtData.vertexD.normal[index.z];

    float3 normal = make_float3(norm4.x,norm4.y,norm4.z);

    float3 ldir = make_float3(0.600,-0.400,0.700);

    const auto &camera=optixLaunchParams.camera;
    const int ix=optixGetLaunchIndex().x;
    const int iy=optixGetLaunchIndex().y;
    const float2 screen(make_float2(ix+.5f,iy+.5f)/make_float2(optixGetLaunchDimensions().x,optixGetLaunchDimensions().y)*2.0-1.0);
    float3 rayDir=normalize(camera.direction+screen.x*camera.horizontal+screen.y*camera.vertical);
    
    // Cor do Reflexo

    float3 reflexoPRD = make_float3(0.f);
    uint32_t u0, u1;
    packPointer( &reflexoPRD, u0, u1 );

    float3 reflexoDir = reflect(rayDir, normal);

    optixTrace(optixLaunchParams.traversable,
        pos,
        reflexoDir,
        0.00001f, // tmin
        1e20f, // tmax
        0.0f, // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_NONE,
        PHONG_RAY_TYPE, // SBT offset
        RAY_TYPE_COUNT, // SBT stride
        PHONG_RAY_TYPE, // missSBTIndex
        u0, u1 );

    // Cor da Transmissão

    float3 transmissaoPRD = make_float3(1.f);
    uint32_t u0_2, u1_2;
    packPointer( &transmissaoPRD, u0_2, u1_2 );

    optixTrace(optixLaunchParams.traversable,
        pos,
        rayDir,
        0.00001f, // tmin
        1e20f, // tmax
        0.0f, // rayTime
        OptixVisibilityMask( 255 ),
        OPTIX_RAY_FLAG_NONE,
        PHONG_RAY_TYPE, // SBT offset
        RAY_TYPE_COUNT, // SBT stride
        PHONG_RAY_TYPE, // missSBTIndex
        u0_2, u1_2 );
    
    // Valor 
    float R0 = 0.04; // pow((1-1.5)/(1+1.5), 2);

    // Schlick's Approximation
    // Para calcular o valor de Schlick's Approximation a usar no PRD final
    float s = R0 + (1 - R0) * pow(1 - abs(dot(normalize(normal), normalize(rayDir))), 5);
    
    prd = transmissaoPRD * (1 - s) + reflexoPRD * s;
}

extern "C" __global__ void __miss__phong_glass() {
    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(1.0f, 1.0f, 1.0f);
}

//GLASS SHADOW FUNCTIONS

// Nothing to do in here
extern "C" __global__ void __anyhit__shadow_glass() {}

extern "C" __global__ void __closesthit__shadow_glass() {
   // get the payload variable
   float3 &prd = *(float3*)getPRD<float3>();

   // get mesh data
   const TriangleMeshSBTData &sbtData
       = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

   // retrieve primitive id and indexes
   const int   primID = optixGetPrimitiveIndex();
   const uint3 index  = sbtData.index[primID];

   // get barycentric coordinates
   const float u = optixGetTriangleBarycentrics().x;
   const float v = optixGetTriangleBarycentrics().y;

   float4 pos4 = (1.f-u-v) * sbtData.vertexD.position[index.x]
       + u * sbtData.vertexD.position[index.y]
       + v * sbtData.vertexD.position[index.z];

   float3 pos = make_float3(pos4.x,pos4.y,pos4.z);

   float4 norm4 = (1.f-u-v) * sbtData.vertexD.normal[index.x]
       + u * sbtData.vertexD.normal[index.y]
       + v * sbtData.vertexD.normal[index.z];

   float3 normal = make_float3(norm4.x,norm4.y,norm4.z);

   float3 ldir = make_float3(0.600,-0.400,0.700);

   const auto &camera=optixLaunchParams.camera;
   const int ix=optixGetLaunchIndex().x;
   const int iy=optixGetLaunchIndex().y;
   const float2 screen(make_float2(ix+.5f,iy+.5f)/make_float2(optixGetLaunchDimensions().x,optixGetLaunchDimensions().y)*2.0-1.0);
   float3 rayDir=normalize(camera.direction+screen.x*camera.horizontal+screen.y*camera.vertical);
   
   // Cor do Reflexo

   float3 PRD = make_float3(0.f);
   uint32_t u0, u1;
   packPointer( &PRD, u0, u1 );

   float3 reflexoDir = reflect(rayDir, normal);

   optixTrace(optixLaunchParams.traversable,
       pos,
       -ldir,
       0.00001f, // tmin
       1e20f, // tmax
       0.0f, // rayTime
       OptixVisibilityMask( 255 ),
       OPTIX_RAY_FLAG_NONE,
       SHADOW_RAY_TYPE, // SBT offset
       RAY_TYPE_COUNT, // SBT stride
       SHADOW_RAY_TYPE, // missSBTIndex
       u0, u1 );

    prd = PRD;
}

extern "C" __global__ void __miss__shadow_glass() {
    float3 &prd = *(float3*)getPRD<float3>();
    // set blue as background color
    prd = make_float3(1.0f, 0.0f, 0.0f);
}

extern "C" __global__ void __raygen__renderFrame() {

    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const auto &camera = optixLaunchParams.camera;  

    // ray payload
    float3 pixelColorPRD = make_float3(1.f);
    uint32_t u0, u1;
    packPointer( &pixelColorPRD, u0, u1 );  

    // compute ray direction
    // normalized screen plane position, in [-1, 1]^2
    const float2 screen(make_float2(ix+.5f,iy+.5f)
            / make_float2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y) * 2.0 - 1.0);

    // note: nau already takes into account the field of view when computing 
    // camera horizontal and vertival
    float3 rayDir = normalize(camera.direction
            + screen.x  * camera.horizontal
            + screen.y * camera.vertical);

    // trace primary ray
    optixTrace(optixLaunchParams.traversable,
            camera.position,
            rayDir,
            0.f,    // tmin
            1e20f,  // tmax
            0.0f,   // rayTime
            OptixVisibilityMask( 255 ),
            OPTIX_RAY_FLAG_NONE,
            PHONG_RAY_TYPE,             // SBT offset
            RAY_TYPE_COUNT,             // SBT stride
            PHONG_RAY_TYPE,             // missSBTIndex 
            u0, u1 );

    //convert float (0-1) to int (0-255)
    const int r = int(255.0f*pixelColorPRD.x);
    const int g = int(255.0f*pixelColorPRD.y);
    const int b = int(255.0f*pixelColorPRD.z);

    // convert to 32-bit rgba value 
    const uint32_t rgba = 0xff000000
        | (r<<0) | (g<<8) | (b<<16);
    // compute index
    const uint32_t fbIndex = ix+iy*optixGetLaunchDimensions().x;
    // write to output buffer
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
}
